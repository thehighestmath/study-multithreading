#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include<chrono>
#include<hip/hip_runtime.h>

__host__ __device__ bool prime(int num) {
	if (num == 0)
		return false;
	for (int i = 2; i <= sqrtf(num); i++) {
		if (num % i == 0) {
			return false;
		}
	}
	return true;
}

bool primeSimple(int num) {
	for (int i = 2; i <= sqrtf(num); i++) {
		if (num % i == 0) {
			return false;
		}
	}
	return true;
}

std::vector<int> decomposition(int number, int count) {
	std::vector<int> summands;
	if (number < count) {
		return summands;
	}

	if (count == 2) {
		int temp = number - count;
		if (primeSimple(number) && primeSimple(temp)) {
			summands.push_back(count);
			summands.push_back(temp);
		}
		return summands;
	}

	if (count >= 3) {
		int remainingSum = number;
		for (int i = 2; i <= remainingSum; i++) {
			if (primeSimple(i)) {
				summands.push_back(i);
				remainingSum -= i;
				count--;
				if (count == 1) {
					summands.push_back(remainingSum);
					return summands;
				}
			}
		}
	}

	throw std::runtime_error("Decomposition not possible for the given number and count.");

	return summands;
}


__global__ void decompositionParallel(int number, int count, int summands[]) {

	if (number < count) {
		return;
	}

	if (count == 2) {
		int temp = number - count;
		if (prime(number) && prime(temp)) {
			summands[0] = count;
			summands[1] = temp;
		}
		return;
	}
	
	if (count >= 3) {
		int remainingSum = number;
		for (int i = 2, j = 0; i <= remainingSum; i++) {
			if (prime(i)) {
				summands[j] = i;
				remainingSum -= i;
				count--;
				if (count == 1) {
					summands[j] = remainingSum;
					return;
				}
				j++;
			}
		}
	}

	//throw std::runtime_error("Decomposition not possible for the given number and count.");return
	return;
}

int main() {
	int N = 0;
	int k = 0;
	int threads = 1;

	std::cout << "Enter N: ";
	std::cin >> N;

	std::cout << "Enter k: ";
	std::cin >> k;

	std::cout << "Enter the numbers of threads: ";
	std::cin >> threads;

	int simpleNum = N + 1;
	while (!prime(simpleNum)) {
		simpleNum++;
	}

	std::cout << "Min prime number: " << simpleNum << std::endl;

	std::vector<int> primers;
	//int *arr = new int[k];

	//Simple
	auto start = std::chrono::high_resolution_clock::now();
	try
	{
		primers = decomposition(simpleNum, k);
	}
	catch (const std::runtime_error& e) {}
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float> duration = end - start;


	//Parallel
	int *dev_summands;
	hipMalloc((void**)&dev_summands, k * sizeof(int));


	int blockSize = 256;
	int numBlocks = (k + blockSize - 1) / blockSize;
	auto startP = std::chrono::high_resolution_clock::now();

	decompositionParallel << <blockSize, numBlocks >> > (simpleNum, k, dev_summands);
	hipDeviceSynchronize();
	auto endP = std::chrono::high_resolution_clock::now();

	hipMemcpy(&primers[0], dev_summands, k * sizeof(int), hipMemcpyDeviceToHost);

	std::chrono::duration<float> durationP = endP - startP;

	hipFree(dev_summands);


	std::cout << std::endl;

	if (primers.empty()) {
		std::cout << "Decomposition is not possible." << std::endl;
	}
	else {
		std::cout << "Summands: ";
		int size = 0;
		k <= 5 ? size = k : size = 5;
		for (int i = 0; i < size; i++) {
			std::cout << primers.at(i) << " ";
		}
		std::cout << std::endl << std::endl;


		/*std::cout << "Arr Result: ";
		for (int i = 0; i < size; i++) {
			std::cout << arr[i] << " ";
		}*/
	}
	std::cout << "Time: " << duration.count() << std::endl;
	std::cout << "Parallel time: " << durationP.count() << std::endl;

	return 0;
}
